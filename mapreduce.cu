/*
 * UCLA Spring 2011
 * CS239
 */


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <unistd.h>

#define NUM_BLOCKS 8
#define BLOCK_SIZE 512

// define map/reduce function type
typedef int (*map_function_t) (int, int);
typedef int (*reduce_function_t) (int, int);

__device__ int rand(int init0, int init1)
{
	// multiply-with-carry RNG
	init0 = 36969 * (init0 & 65535) + (init0 >> 16);
	init1 = 18000 * (init1 & 65535) + (init1 >> 16);
	return (init0 << 16) + init1;	/* 32-bit result */
}

__device__ int fma0(int op0, int op1)
{
	return op0 + op0 * op1;
}

__device__ int sum(int op0, int op1)
{
	return op0 + op1;
}

// NVIDIA reference implementation
template < unsigned int blockSize >
    __global__ void reduce(int *g_idata, int *g_odata, unsigned int n)
{
	extern __shared__ int sdata[];
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * (blockSize * 2) + tid;
	unsigned int gridSize = blockSize * 2 * gridDim.x;
	sdata[tid] = 0;

	while (i < n) {
		sdata[tid] += g_idata[i] + g_idata[i + blockSize];
		i += gridSize;
	}
	__syncthreads();
	if (blockSize >= 512) {
		if (tid < 256) {
			sdata[tid] += sdata[tid + 256];
		}
		__syncthreads();
	}
	if (blockSize >= 256) {
		if (tid < 128) {
			sdata[tid] += sdata[tid + 128];
		}
		__syncthreads();
	}
	if (blockSize >= 128) {
		if (tid < 64) {
			sdata[tid] += sdata[tid + 64];
		}
		__syncthreads();
	}

	if (tid < 32) {
		if (blockSize >= 64)
			sdata[tid] += sdata[tid + 32];
		if (blockSize >= 32)
			sdata[tid] += sdata[tid + 16];
		if (blockSize >= 16)
			sdata[tid] += sdata[tid + 8];
		if (blockSize >= 8)
			sdata[tid] += sdata[tid + 4];
		if (blockSize >= 4)
			sdata[tid] += sdata[tid + 2];
		if (blockSize >= 2)
			sdata[tid] += sdata[tid + 1];
	}

	if (tid == 0) {
		g_odata[blockIdx.x] = sdata[0];
	}
}

__global__ void mapreduce(int *array, int count, int *g_cache, int *result)
{
	// gridDim.x is number of blocks
	// blockDim.x is number of threads per block
	// b_cache should be equally sized to blockDim.x
	// g_cache should be equally sized to gridDim.x
	extern __shared__ int b_cache[];

	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int i, thread_work_size, thread_offset;

	if (tid == gridDim.x * blockDim.x - 1) {
		int normal_work_size = count / (gridDim.x * blockDim.x);
		thread_offset = tid * normal_work_size;
		thread_work_size = count - thread_offset;
	} else {
		thread_work_size = count / (gridDim.x * blockDim.x);
		if (thread_work_size < 1) {
			thread_work_size = 1;
		}
		thread_offset = tid * thread_work_size;
	}

	// map section
	b_cache[threadIdx.x] = array[thread_offset];
	for (i = thread_offset + 1; i < thread_offset + thread_work_size; i++) {
		b_cache[threadIdx.x] = rand(b_cache[threadIdx.x], array[i]);
	}
	__syncthreads();

	// reduce section

	// get the largest number in a block
	if (threadIdx.x == 0) {
		int largest = b_cache[0];
		for (i = 1; i < blockDim.x; i++) {
			if (b_cache[i] > largest) {
				largest = b_cache[i];
			}
		}
		g_cache[blockIdx.x] = largest;
	}
	__syncthreads();

	// get the largest number in all blocks
	if (tid == 0) {
		int largest = g_cache[0];
		for (i = 1; i < gridDim.x; i++) {
			if (g_cache[i] > largest) {
				largest = g_cache[i];
			}
		}
		*result = largest;
	}
}

void usage(int which)
{
	switch (which) {
	default:
		printf("usage: mapreduce [-b blocks|-t threads] <filename>\n");
		break;
	case 1:
		printf("mapreduce input format:\nnum count\n1\n...\nn n\n");
		break;
	case 2:
		printf("mapreduce requires numbers >= threads*blocks\n");
		break;
	}
}

int prepare_numbers(const char *filename, int **array)
{
	int count, input, i;
	FILE *file;

	file = fopen(filename, "r");

	// count of data is first line
	fscanf(file, "%d", &count);
	int *numbers = (int *)malloc(count * sizeof(int));

	// load array
	for (i = 0; i < count; i++) {
		if (fscanf(file, "%d", &input) < 0) {
			break;
		}
		numbers[i] = input;
	}
	fclose(file);

	if (count != i) {
		free(numbers);
		return -1;
	} else {
		*array = numbers;
		return count;
	}
}

int main(int argc, char *argv[])
{
	int opt, blocks, threads, array_size, result;
	int *array_h, *array_d, *result_d, *cache_d;
	char *filename;

	// set options
	blocks = NUM_BLOCKS;
	threads = BLOCK_SIZE;
	while ((opt = getopt(argc, argv, "b:t:")) != -1) {
		switch (opt) {
		case 'b':
			blocks = atoi(optarg);
			break;
		case 't':
			threads = atoi(optarg);
			break;
		default:
			usage(0);
			return 0;
		}
	}
	dim3 dim_grid(blocks, 1, 1);
	dim3 dim_block(threads, 1, 1);

	// check to make sure we are feeding in correct number of args
	if (argc == optind + 1) {
		filename = argv[optind];
	} else {
		usage(0);
		return 0;
	}

	// read file
	array_h = NULL;
	array_size = prepare_numbers(filename, &array_h);
	if (array_size < 0) {
		free(array_h);
		usage(1);
		return 0;
	} else if (array_size <= blocks * threads) {
		free(array_h);
		usage(2);
		return 0;
	}

	result = 0;
	printf("mapreduce using CUDA\n");
	// move to device
	hipMalloc((void **)&array_d, array_size * sizeof(int));
	hipMemcpy(array_d, array_h, array_size * sizeof(int),
		   hipMemcpyHostToDevice);

	// allocate device only structures
	hipMalloc((void **)&result_d, sizeof(int));
	hipMalloc((void **)&cache_d, blocks * sizeof(int));

	// run kernel
	mapreduce <<< dim_grid, dim_block, threads * sizeof(int) >>> (array_d,
								      array_size,
								      cache_d,
								      result_d);

	// retrieve result
	hipMemcpy(&result, result_d, sizeof(int), hipMemcpyDeviceToHost);

	// cleanup
	hipFree(array_d);
	hipFree(result_d);
	hipFree(cache_d);

	printf("mapreduce result: %d\n", result);
	free(array_h);

	return 0;
}
